#include "annfab/common.hpp"
#include "annfab/math_functions.hpp"
#include "annfab/annfab_cuda_utils.hpp"

namespace annfab {

template <>
void annfab_gpu_gemm<float>(hipblasHandle_t handle, const hipblasOperation_t TransA,
    const hipblasOperation_t TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == HIPBLAS_OP_N) ? K : M;
  int ldb = (TransB == HIPBLAS_OP_N) ? N : K;
  assert_on_cuda_error(hipblasSgemm(handle, TransB, TransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void annfab_gpu_gemm<double>(hipblasHandle_t handle, const hipblasOperation_t TransA,
    const hipblasOperation_t TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == HIPBLAS_OP_N) ? K : M;
  int ldb = (TransB == HIPBLAS_OP_N) ? N : K;
  assert_on_cuda_error(hipblasDgemm(handle, TransB, TransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

}  // name